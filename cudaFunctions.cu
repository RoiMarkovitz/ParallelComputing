#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "cFunctions.h"

                             
__global__ void calculateAlignmentScores(char* dev_seq1, char* dev_seq2, int seq2_len, int* dev_weights, AlignmentScore* dev_score_array, int current_location_in_score_array, int* dev_signs_match_matrix, int max_threads_per_block)
{    
    // each block calculates a score for a pair of offset and mutant_lvl 
    int score_index = blockIdx.y * gridDim.x + blockIdx.x + current_location_in_score_array; 
    int thread_local_id = threadIdx.x;
    int skip_letter = 0;
    int cycle = 0; // number of cycles of max_threads_per_block
    int i,j;
     
    // max num of compares is seq2_len 
    if (thread_local_id < seq2_len) 
    { 	
        do
        {
            // in the grid -> blockIdx.y + 1 = mutant_lvl = k
		    if (thread_local_id >= blockIdx.y + 1) // ignore "-" comparison                
			    skip_letter = 1;

            i = dev_seq1[thread_local_id+blockIdx.x+skip_letter] - 'A'; 
            j = dev_seq2[thread_local_id] - 'A';
            atomicAdd(&dev_score_array[score_index].score, dev_weights[dev_signs_match_matrix[i*NUM_LETTERS + j]]);
            cycle++;
            thread_local_id = threadIdx.x + cycle*max_threads_per_block;
        } while (thread_local_id < seq2_len); // check if thread should work again     
    }
             
    if(threadIdx.x == 0) 
    {
        dev_score_array[score_index].offset = blockIdx.x; 
        dev_score_array[score_index].mutant_lvl = blockIdx.y + 1; // blockIdx.y + 1 = mutant_lvl of the block in the grid. 
    }

}

int checkStatus(hipError_t cudaStatus, char* dev_seq1, char* dev_seq2, AlignmentScore* dev_score_array, int* dev_weights, int* dev_signs_match_matrix, std::string err)
{
    if(cudaStatus != hipSuccess)
    {
        std::cout << err <<std::endl;

        if (dev_seq1 != NULL)
            free(dev_seq1);
        if (dev_seq2 != NULL)      
            free(dev_seq2);
        if (dev_score_array != NULL) 
            free(dev_score_array);
        free(dev_weights);
        free(dev_signs_match_matrix);
           
        return 1;
    }
    return 0; // all is fine
}

 
int startCudaAlignmentScoreCalculation(char** work_arr, int work_arr_size, char* seq1, int* weights, AlignmentScore* score_array, int* score_array_sizes, int score_array_total_size)
{
    char* dev_seq1 = NULL;  
    char* dev_seq2 = NULL; 
    AlignmentScore* dev_score_array = NULL;
    int seq1_len, seq2_len;
    int* dev_weights = 0;
    int* dev_signs_match_matrix = 0;
    int size, offsets, mutants;
    int max_threads_per_block;
    int current_location_in_score_array = 0;
  
    hipError_t cudaStatus;

    hipDeviceProp_t prop;
    cudaStatus = hipGetDeviceProperties(&prop,0);

    // SIGNS_MATCH_MATRIX allocation and copy from host to device
    size = NUM_LETTERS * NUM_LETTERS * sizeof(int);
    cudaStatus = hipMalloc((void**)&dev_signs_match_matrix, size);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda malloc for dev_signs_match_matrix failed!"))
        return 1;
    cudaStatus = hipMemcpy(dev_signs_match_matrix, SIGNS_MATCH_MATRIX, size, hipMemcpyHostToDevice);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy SIGNS_MATCH_MATRIX to device failed!"))
        return 1; 
    
    // weights memory allocation and copy from host to device
    size = sizeof(int)*NUM_SIGNS;
    cudaStatus = hipMalloc((void**)&dev_weights, size);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda malloc for dev_weights failed!"))
        return 1;
    cudaStatus = hipMemcpy(dev_weights, weights, size, hipMemcpyHostToDevice);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy weights to device failed!"))
        return 1;

    // seq1 memory allocation and copy from host to device
    seq1_len = strlen(seq1);
    size = sizeof(char)* (seq1_len + 1);
    cudaStatus = hipMalloc((void**)&dev_seq1, size);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda malloc for dev_seq1 failed!"))
        return 1;
    cudaStatus = hipMemcpy(dev_seq1, seq1, size, hipMemcpyHostToDevice);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy seq1 to device failed!"))
        return 1;

    // dev_score_array memory allocation and copy from host to device
    size = sizeof(AlignmentScore) * score_array_total_size;
    cudaStatus = hipMalloc((void**)&dev_score_array, size);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda malloc for dev_score_array failed!"))
        return 1;
    cudaStatus = hipMemcpy(dev_score_array, score_array, size, hipMemcpyHostToDevice);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy score_array to device failed!"))
        return 1;
    
    // loops over seq2 strings
    for (int i = 0; i < work_arr_size; i++)
    {
        seq2_len = strlen(work_arr[i]); 
        size = sizeof(char)* (seq2_len + 1);   
        cudaStatus = hipMalloc((void**)&dev_seq2, size);
        if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda malloc for dev_seq2 failed!"))
            return 1;  
        cudaStatus = hipMemcpy(dev_seq2, work_arr[i], size, hipMemcpyHostToDevice);
        if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy work_arr[i] to device failed!"))
            return 1;
    
        offsets = seq1_len - seq2_len;
        mutants = seq2_len; 
        dim3 numBlocks (offsets, mutants);
        max_threads_per_block = prop.maxThreadsPerBlock < mutants ? prop.maxThreadsPerBlock : mutants;
      
        // activate kernel
        calculateAlignmentScores<<<numBlocks,max_threads_per_block>>>(dev_seq1, dev_seq2, seq2_len, dev_weights, dev_score_array, current_location_in_score_array, dev_signs_match_matrix, max_threads_per_block);
        cudaStatus = hipDeviceSynchronize();
        if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda Kernel for calculateAlignmentScores failed!"))
            return 1;
        
        current_location_in_score_array += score_array_sizes[i];
    }
  
    // copy data from device to host
    size = sizeof(AlignmentScore) * score_array_total_size;
    cudaStatus = hipMemcpy(score_array, dev_score_array, size, hipMemcpyDeviceToHost);
    if(checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda memcpy dev_score_array to host failed!"))
        return 1;

	// free cude memory   
    cudaStatus = hipFree(dev_signs_match_matrix);
    if (checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda free dev_signs_match_matrix failed!"))
       return 1;
    cudaStatus = hipFree(dev_weights);
    if (checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda free dev_weights failed!"))
        return 1;
    cudaStatus = hipFree(dev_seq1);
    if (checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda free dev_seq1 failed!"))
        return 1;    
    cudaStatus = hipFree(dev_seq2);
    if (checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda free dev_seq2 failed!"))
        return 1;   
    cudaStatus = hipFree(dev_score_array);
    if (checkStatus(cudaStatus, dev_seq1, dev_seq2, dev_score_array, dev_weights, dev_signs_match_matrix, "Cuda free dev_score_array failed!"))
        return 1;
 
    return 0; // all is fine
}


